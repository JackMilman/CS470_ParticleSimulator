#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <random>
#include <cstdlib>
#include <stack>
#include <unistd.h>

#include <GL/glew.h>
#include <GL/freeglut.h>

#include "particle.cuh"
#include "particle.cu"
#include "vector.cuh"
#include "vector.cu"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

int num_particles;
float particle_size;
Particle* particles;
Particle* device_particles;
hiprandState* states;

int lastTime;

// GL functionality
bool initGL(int *argc, char **argv);

// Check for collisions and resolve them
__global__ void checkCollision(Particle* d_particles, int n_particles) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    for (int j = i + 1; j < n_particles; j++) {
        if (d_particles[i].collidesWith(d_particles[j])) {
            d_particles[i].resolveCollision(d_particles[j]);
        }
    }
}

// Update the position of the particles and check for wall collisions
__global__ void updateParticles(Particle* d_particles, int n_particles, hiprandState* states, float deltaTime) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n_particles) {
        d_particles[i].updatePosition(deltaTime);
        d_particles[i].wallBounce();
    }
}

// Host function
void display() {

	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    // Render particles
    for (int i = 0; i < num_particles; i++) {
        particles[i].renderSphere();
    }

    int blockSize = 256;
    int blockCount = (num_particles + blockSize - 1) / blockSize;

    // FPS counter
    static int frameCount = 0;
    int currentTime = glutGet(GLUT_ELAPSED_TIME);
    float delta = (currentTime - lastTime) / 1000.0f;
    lastTime = currentTime;
    frameCount++;

    if (frameCount % 20 == 0) {
        char title[80];
        sprintf(title, "Particle Simulator (%.2f fps) - %d particles", 1 / delta, num_particles);
        printf("%f\n", 1 / delta);
        glutSetWindowTitle(title);
    }

    // Send particle data to device
    hipMemcpy(device_particles, particles, num_particles * sizeof(Particle), hipMemcpyHostToDevice);
    updateParticles<<<blockCount, blockSize>>>(device_particles, num_particles, states, delta);
    checkCollision<<<blockCount, blockSize>>>(device_particles, num_particles);
    // Retrieve particle data from device
    hipMemcpy(particles, device_particles, num_particles * sizeof(Particle), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    glutSwapBuffers();
}

void timer( int value )
{
    glutPostRedisplay();
    glutTimerFunc( 16, timer, 0 );
}

bool initGL(int *argc, char **argv)
{
    glutInit(argc, argv);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA | GLUT_DEPTH);
    glutInitWindowSize(800, 800);
    glutCreateWindow("3D Particle Simulator");
    glutPositionWindow(950,100);
    glutTimerFunc( 0, timer, 0 );
    glutDisplayFunc(display);

    glewExperimental = GL_TRUE;
    GLenum err = glewInit();
    if (err != GLEW_OK) {
        fprintf(stderr, "GLEW initialization failed: %s\n", glewGetErrorString(err));
        return false;
    }

    glEnable(GL_DEPTH_TEST);
    glDepthFunc(GL_LESS);

    glClearColor(0.1f, 0.1f, 0.1f, 1.0f);

    // Setup perspective projection
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(60.0, 1.0, 0.1, 10.0);

    // Setup the camera
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    gluLookAt(1000.0, 0.0, 1000.0,
              0.0, 0.0, 0.0,
              0.0, 1.0, 0.0);

    return true;
    // glutInit(argc, argv);
    // glutInitWindowSize(800, 800);
    // glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
    // glutCreateWindow("Particle Simulator");
    // glutPositionWindow(950,100);
    // glutTimerFunc( 0, timer, 0 );
    // glutDisplayFunc(display);

    // // Initialize GLEW
    // glewExperimental = GL_TRUE;
    // GLenum err = glewInit();
    // if (err != GLEW_OK) {
    //     fprintf(stderr, "GLEW initialization failed: %s\n", glewGetErrorString(err));
    //     return false;
    // }

    // return true;
}

int main(int argc, char** argv) {
    // Set defaults
    srand(time(NULL));
    num_particles = 100;
    particle_size = 0.1f;
    int opt;
    bool explode = false;

    // Command line options
    while ((opt = getopt(argc, argv, "n:s:e")) != -1) {
        switch (opt) {
            case 'n':
                num_particles = strtol(optarg, NULL, 10);
                break;
            case 's':
                particle_size = strtod(optarg, NULL);
                break;
            case 'e':
                // Explode particles from center. Recommend running with a lot of particles with a low size
                explode = true;
                break;
            default:
                fprintf(stderr, "Usage: %s [-n num_particles] [-sp particle_size] [-e explosion (OPTIONAL)]\n", argv[0]);
                exit(EXIT_FAILURE);
        }
    }

    particles = (Particle*) calloc(num_particles, sizeof(Particle));

    for (int i = 0; i < num_particles; i++) {
        std::random_device rd;
        std::mt19937 gen(rd());

        // Randomize velocity and position in 3D
        std::uniform_real_distribution<float> dist(-2, 2);
        std::uniform_real_distribution<float> randPosition(-0.95, 0.95);
        std::uniform_real_distribution<float> mass(1.5, 5);

        float dx = dist(gen);
        float dy = dist(gen);
        float dz = dist(gen);  // z-velocity

        float x, y, z;
        if (explode) {
            x = y = z = 0;  // Explode from center
        } else {
            x = randPosition(gen);
            y = randPosition(gen);
            z = randPosition(gen);  // z-coordinate
        }

        particles[i] = Particle(Vector(x, y, z), Vector(dx, dy, dz), mass(gen), particle_size);
    }


    // Init the device particles
    hipMalloc((void**)&device_particles, num_particles * sizeof(Particle));
    hipMalloc((void**)&states, num_particles * sizeof(hiprandState));

    initGL(&argc, argv);

    lastTime = 0;
    glutMainLoop();

    // Clean up
    hipDeviceSynchronize();
    hipFree(device_particles);
    hipFree(states);

    return 0;
}