#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <random>
#include <cstdlib>
#include <stack>
#include <unistd.h>
#include <set>
#include <unordered_set>
#include <chrono>
#include <iomanip>

#include <GL/glew.h>
#include <GL/freeglut.h>

#include "particle.cuh"
#include "particle.cu"
#include "vector.cuh"
#include "vector.cu"
#include "edge.cu"
#include "edge.cuh"
#include "particle_pair.cu"
#include "particle_pair.cuh"
#include "quadtree.cu"
// #include "quadtree.cuh"
#include "spatial_hashing.cu"
#include "spatial_hashing.cuh"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define DEFAULT_P_SIZE 0.05f
#define DEFAULT_P_NUMBER 50
#define PI 3.14159265f
#define NUM_CMD "-n num_particles"
#define SIZE_CMD "-s particle_size"
#define EXPLODE_CMD "-e explode_from_center"
#define SWEEP_CMD "-w sweep_and_prune"
#define QUAD_CMD "-t quad_tree"
#define SPATIAL_CMD "-g spatial_hash"
#define HELP_CMD "-h help"

int num_particles;
float particle_size;
Particle* particles;
Particle* device_particles;

enum modes {BruteForce, SweepAndPrune, Quad, Hash};
int mode = BruteForce;

// Rectangle rectangle = Rectangle((float) X_MIN, (float) Y_MIN, (float) X_MAX, (float) Y_MAX);
// QuadTree quadtree = QuadTree(0, rectangle);
Edge* edgesByX;
int num_edges;
int max_pairs;
std::unordered_set<int>* p_overlaps;
std::unordered_set<int>* device_overlaps;
ParticlePair* pairs;
ParticlePair* device_pairs;

float cellSize = DEFAULT_P_SIZE;
SpatialHash spatialHash(cellSize);

int lastTime;

// Testing variables
std::chrono::duration<double, std::milli> cumulativeTime(0);
unsigned long long bruteForceOps = 0;
unsigned long long sweepAndPruneOps = 0;
unsigned long long spatialHashOps = 0;
unsigned long long treeOps = 0;
std::chrono::duration<double> bruteForceTime(0);
std::chrono::duration<double> sweepAndPruneTime(0);
std::chrono::duration<double> spatialHashTime(0);

// GL functionality
bool initGL(int *argc, char **argv);

void sortByX(Edge* edges) {
    // Simple insertion sort for the particles, sorting by their x-positions. This is to be used in sweep-and-prune.
    for (int i = 1; i < num_edges; i++) {
        for (int j = i - 1; j >= 0; j--) {
            Particle& p_j = particles[edges[j].getParentIdx()];
            Particle& p_next_j = particles[edges[j + 1].getParentIdx()];

            bool j_left = edges[j].getIsLeft();
            float j_x = j_left ? p_j.getPosition().getX() - particle_size: p_j.getPosition().getX() + particle_size;

            bool j_next_left = edges[j + 1].getIsLeft();
            float j_next_x = j_next_left ? p_next_j.getPosition().getX() - particle_size: p_next_j.getPosition().getX() + particle_size;

            if (j_x < j_next_x) break;
            Edge tmp = edges[j];
            edges[j] = edges[j + 1];
            edges[j + 1] = tmp;
        }
    }
}

// A simple check to determine if a particle pair has already been added to our overlap tracker.
bool resolved(int p_edge, int other) {
    bool resolved = p_overlaps[p_edge].count(other) == 1;
    return resolved;
}

/* Sweeps across the list of particle edges, sorted by their minimum x-values. 
   If an edge is a left-edge, we look at all the other particles currently
   being "touched" by our imaginary line and check if they have already been
   resolved. If they have not yet been resolved, we perform a finer-grained
   check to see if they collide, and resolve a collision if they do. */
int sweepAndPruneByX() {
    sortByX(edgesByX);
    // indexes of particles touched by the line at this point
    std::unordered_set<int> touching;
    int p_edge_idx;
    // pair_idx represents the number of overlaps (minus one) that we have found in
    // this iteration. It will be used to determine how far into the pairs array to
    // look, when fine-checking for collisions
    int pair_idx = 0;
    for (int i = 0; i < num_edges; i++) {
        p_edge_idx = edgesByX[i].getParentIdx();
        if (edgesByX[i].getIsLeft()) {
            for (auto itr = touching.begin(); itr != touching.end(); ++itr) {
                    pairs[pair_idx] = ParticlePair(p_edge_idx, *itr);
                    pair_idx++;
            }
            touching.insert(p_edge_idx);
        } else {
            touching.erase(p_edge_idx);
        }
    }
    return pair_idx - 1;
}


// Check for collisions and resolve them
__global__ void checkBruteForce(Particle* d_particles, int n_particles) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    for (int j = i + 1; j < n_particles; j++) {
        if ((i != j) && d_particles[i].collidesWith(d_particles[j])) {
            d_particles[i].resolveCollision(d_particles[j]);
        }
    }
}

__global__ void checkSweep(Particle* d_particles, ParticlePair* d_pairs, int n_pairs) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n_pairs; i+=stride) {
        Particle& a = d_particles[d_pairs[i].getA()];
        Particle& b = d_particles[d_pairs[i].getB()];
        if (a.collidesWith(b)) {
            a.resolveCollision(b);
        }
    }
}

// Update the position of the particles and check for wall collisions
__global__ void updateParticles(Particle* d_particles, int n_particles, float deltaTime) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n_particles) {
        d_particles[i].updatePosition(deltaTime);
        d_particles[i].wallBounce();
    }
}

// Host function
void display() {
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    // FPS counter
    static int frameCount = 0;
    int currentTime = glutGet(GLUT_ELAPSED_TIME);
    float delta = (currentTime - lastTime) / 1000.0f;
    lastTime = currentTime;
    frameCount++;

    // Render particles
    for (int i = 0; i < num_particles; i++) {
        particles[i].render();
    }

    int blockSize = 256;
    int blockCount = (num_particles + blockSize - 1) / blockSize;

    if (frameCount == 1000) {
        double averageTime = cumulativeTime.count() / frameCount;
        std::cout << "Average time per frame: " 
              << std::fixed << std::setprecision(10) 
              << averageTime << " ms" << std::endl;
        switch (mode) {
            case BruteForce:
                std::cout << "Brute Force Ops: " << bruteForceOps << "\n";
                break;
            case SweepAndPrune:
                std::cout << "Sweep and Prune Ops: " << sweepAndPruneOps << "\n";
                break;
            case Quad:
                std::cout << "Quadtree Ops: " << treeOps << "\n";
                break;
            case Hash:
                std::cout << "Spatial Hash Ops: " << spatialHashOps << "\n";
                break;
            default:
                break;
        }
        exit(EXIT_SUCCESS);
    }

    if (frameCount % 20 == 0) {
        char title[80];
        sprintf(title, "Particle Simulator (%.2f fps) - %d particles", 1 / delta, num_particles);
        // printf("%f\n", 1 / delta);
        glutSetWindowTitle(title);
    }

    int num_ops = 0;
    int n_pairs = 0;

    int storageSize = 0;
    int* d_storageSize;
    hipMalloc((void**)&d_storageSize, sizeof(int));
    hipMemcpy(d_storageSize, &storageSize, sizeof(int), hipMemcpyHostToDevice);

    int* d_keys, *d_particleIndices;
    hipMalloc((void**)&d_keys, num_particles * sizeof(int));
    hipMalloc((void**)&d_particleIndices, num_particles * sizeof(int));

    // Send particle data to device
    hipMemcpy(device_particles, particles, num_particles * sizeof(Particle), hipMemcpyHostToDevice);
    updateParticles<<<blockCount, blockSize>>>(device_particles, num_particles, delta);
    hipDeviceSynchronize();
    auto start = std::chrono::high_resolution_clock::now();
    switch (mode) {
        case BruteForce:
            checkBruteForce<<<blockCount, blockSize>>>(device_particles, num_particles);
            break;
        case SweepAndPrune:
            n_pairs = sweepAndPruneByX();
            hipMemcpy(device_pairs, pairs, max_pairs * sizeof(ParticlePair), hipMemcpyHostToDevice);
            checkSweep<<<blockCount, blockSize>>>(device_particles, device_pairs, n_pairs);
            hipMemcpy(pairs, device_pairs, max_pairs * sizeof(ParticlePair), hipMemcpyDeviceToHost);
            break;
        case Quad:
            ////////////    QuadTree Implementation - See analysis for explanation on CPU-GPU dependencies      ////////////
            break;
        case Hash:
            spatialHash.clear();
            insertParticles<<<blockCount, blockSize>>>(device_particles, num_particles, cellSize, d_storageSize, d_keys, d_particleIndices);
            hipDeviceSynchronize();
            queryParticles<<<blockCount, blockSize>>>(device_particles, num_particles, cellSize, d_storageSize, d_keys, d_particleIndices);
            break;
    }
    hipMemcpy(&storageSize, d_storageSize, sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    // Retrieve particle data from device
    hipMemcpy(particles, device_particles, num_particles * sizeof(Particle), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    cumulativeTime += end - start;
    switch (mode) {
        case BruteForce:
            bruteForceOps += num_ops;
            break;
        case SweepAndPrune:
            sweepAndPruneOps += num_ops;
            break;
        case Quad:
            treeOps += num_ops;
            break;
        case Hash:
            spatialHashOps += num_ops;
            break;
    }

    hipFree(d_keys);
    hipFree(d_particleIndices);
    hipFree(d_storageSize);

    glutSwapBuffers();
}

void timer( int value )
{
    glutPostRedisplay();
    glutTimerFunc( 16, timer, 0 );
}

bool initGL(int *argc, char **argv)
{
    glutInit(argc, argv);
    glutInitWindowSize(800, 800);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA | GLUT_DEPTH);
    glutCreateWindow("3D Particle Simulator");
    glutPositionWindow(950,100);
    glutTimerFunc( 0, timer, 0 );
    glutDisplayFunc(display);

    glewExperimental = GL_TRUE;
    GLenum err = glewInit();
    if (err != GLEW_OK) {
        fprintf(stderr, "GLEW initialization failed: %s\n", glewGetErrorString(err));
        return false;
    }

    return true;
}

bool good_args(int argc, char** argv, bool* explode) {
     // Command line options
    int opt;
    while ((opt = getopt(argc, argv, "n:s:ewhtg")) != -1) {
        switch (opt) {
            case 'n':
                num_particles = strtol(optarg, NULL, 10);
                break;
            case 's':
                particle_size = strtod(optarg, NULL);
                break;
            case 'e':
                // Explode particles from center. Recommend running with a lot of particles with a low size
                *explode = true;
                break;
            case 'w':
                if (mode != BruteForce)
                    return false;
                mode = SweepAndPrune;
                break;
            case 't':
                if (mode != BruteForce)
                    return false;
                mode = Quad;
                break;
            case 'g':
                if (mode != BruteForce)
                    return false;
                mode = Hash;
                break;
            case 'h':
                return false;
                break;
            default:
                return false;
                break;
        }
        switch(mode) {
            case BruteForce:
                break;
            case SweepAndPrune:
                num_edges = num_particles * 2;
                edgesByX = (Edge*) calloc(num_edges, sizeof(Edge));
                p_overlaps = new std::unordered_set<int>[num_particles];
                break;
            case Quad:
                // rectangle = new Rectangle((float) X_MIN, (float) Y_MIN, (float) X_MAX, (float) Y_MAX);
                // quadtree = new QuadTree(0, *rectangle);
                break;
        }
    }
    return true;
}

int main(int argc, char** argv) {
    // Set defaults
    srand(time(NULL));
    num_particles = DEFAULT_P_NUMBER;
    particle_size = DEFAULT_P_SIZE;
    bool explode = false;

    if (!good_args(argc, argv, &explode)) {
        fprintf(stderr, "Usage: %s [%s] [%s] [%s (OPTIONAL)] [%s | %s | %s (OPTIONAL)]\n", argv[0],
            NUM_CMD, SIZE_CMD, EXPLODE_CMD, SWEEP_CMD, QUAD_CMD, SPATIAL_CMD);
        exit(EXIT_FAILURE);
    }

    particles = (Particle*) calloc(num_particles, sizeof(Particle));
    // num_edges = num_particles * 2;
    // edgesByX = (Edge*) calloc(num_edges, sizeof(Edge));
    // p_overlaps = new std::unordered_set<int>[num_particles];
    // max_pairs = num_particles * num_particles;
    // pairs = (ParticlePair*) calloc(max_pairs, sizeof(ParticlePair));


    for (int i = 0; i < num_particles; i++) {
        std::random_device rd;
        std::mt19937 gen(rd());

        // Randomize velocity, position, depth, and mass
        std::uniform_real_distribution<float> velocity(VEL_MIN, VEL_MAX);
        std::uniform_real_distribution<float> position_x(X_MIN + particle_size, X_MAX - particle_size);
        std::uniform_real_distribution<float> position_y(Y_MIN + particle_size, Y_MAX - particle_size);
        std::uniform_real_distribution<float> mass(1.5, 5);

        float dx = velocity(gen);
        float dy = velocity(gen);

        float x, y;
        if (explode) {
            x = (X_MAX + X_MIN) / 2;
            y = (Y_MAX + Y_MIN) / 2;
        } else {
            x = position_x(gen);
            y = position_y(gen);
        }

        particles[i] = Particle(Vector(x, y), Vector(dx, dy), mass(gen), particle_size);
    }
    
    // Init the device particles
    hipMalloc((void**)&device_particles, num_particles * sizeof(Particle));
    switch (mode) {
        case SweepAndPrune:
            for (int i = 0; i < num_particles; i++) {
                edgesByX[i*2] = Edge(i, false);
                edgesByX[i*2 + 1] = Edge(i, true);
            }
            sortByX(edgesByX);
            hipMalloc((void**)&device_overlaps, num_particles * sizeof(p_overlaps));
            hipMalloc((void**)&device_pairs, max_pairs * sizeof(ParticlePair));
            break;
        case Hash:
            SpatialHash* spatialHash;
            hipMalloc(&spatialHash, sizeof(SpatialHash));
            float cellSize = 2 * particle_size;  // Cell size can be twice the particle size
            SpatialHash newHash(cellSize);
            hipMemcpy(spatialHash, &newHash, sizeof(SpatialHash), hipMemcpyHostToDevice);
            break;
    }
    

    

    
    initGL(&argc, argv);

    lastTime = 0;
    glutMainLoop();

    // Clean up
    hipDeviceSynchronize();
    hipFree(device_particles);
    hipFree(device_pairs);

    return 0;
}